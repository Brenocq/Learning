// nvcc matrixMulti.cu -o a && ./a

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define WIDTH 3

__global__ void matrixMulKernelSimple(float* M, float* N, float* P, int width)
{
	// Simple kernel, accessing global memory too much! (Probabily using less than 1% of the total GPU power due to limited memory communication bandwidth)
	int col = blockDim.x*blockIdx.x + threadIdx.x;
	int row = blockDim.y*blockIdx.y + threadIdx.y;

	float res=0;
	if(col<width && row<width)
	{
		for(int i=0; i<width; i++)
			res += M[row*width+i]*N[i*width+col];

		P[row*width+col] = res;
	}
}

int main()
{
	float *d_M, *d_N, *d_P;
	float *h_M, *h_N, *h_P;

	int id = hipGetDevice(&id);
	printf("GPU id: %d\n",id);

	// Alloc memory CPU
	h_M = (float*)malloc(WIDTH*WIDTH*sizeof(float));
	h_N = (float*)malloc(WIDTH*WIDTH*sizeof(float));
	h_P = (float*)malloc(WIDTH*WIDTH*sizeof(float));

	// Alloc memory GPU
	hipMalloc((void**)&d_M, WIDTH*WIDTH*sizeof(float));
	hipMalloc((void**)&d_N, WIDTH*WIDTH*sizeof(float));
	hipMalloc((void**)&d_P, WIDTH*WIDTH*sizeof(float));

	// Populate matrices
	for(int i=0;i<WIDTH*WIDTH;i++)
	{
		h_M[i]=i;
		h_N[i]=2*i;
		h_P[i]=0;
	}

	// Copy from CPU to GPU
	hipMemcpy(d_M, h_M, WIDTH*WIDTH*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_N, h_N, WIDTH*WIDTH*sizeof(float), hipMemcpyHostToDevice);

	dim3 gridDim3 = dim3(1,1,1);
	dim3 blockDim3 = dim3(ceil(WIDTH/32.0f)*32, ceil(WIDTH/32.0f)*32,1);
	matrixMulKernelSimple<<<gridDim3, blockDim3>>>(d_M,d_N,d_P, WIDTH);
	
	// Copy from GPU to CPU
	hipMemcpy(h_P, d_P, WIDTH*WIDTH*sizeof(float), hipMemcpyDeviceToHost);

	// Print
	for(int i=0;i<WIDTH;i++)
	{
		for(int j=0;j<WIDTH;j++)
			printf("%c %f%c",j==0?'[':'\t',h_M[i*WIDTH+j], j==WIDTH-1?']':' ');

		printf("\t");

		for(int j=0;j<WIDTH;j++)
			printf("%c %f%c",j==0?'[':'\t',h_N[i*WIDTH+j], j==WIDTH-1?']':' ');

		printf("\t");

		for(int j=0;j<WIDTH;j++)
			printf("%c %f%c",j==0?'[':'\t',h_P[i*WIDTH+j], j==WIDTH-1?']':' ');
		printf("\n");
	}

	// Free
	hipFree(d_M);
	hipFree(d_N);
	hipFree(d_P);
	free(h_M);
	free(h_N);
	free(h_P);

	return 0;
}
