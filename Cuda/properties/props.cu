// nvcc props.cu -o a && ./a
#include <stdio.h>

int main()
{
	int devCount;
	hipGetDeviceCount(&devCount);
	printf("Found %d devices with CUDA support.\n", devCount);

	for(int i=0; i<devCount; i++)
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, i);
		printf("- DEVICE %d:\n",i);
		printf("  - Name: %s\n", devProp.name);
		printf("  - Max threads per block: %d\n", devProp.maxThreadsPerBlock);
		printf("  - Max blocks per SM: %d\n", devProp.maxBlocksPerMultiProcessor);
		printf("  - Qty SMs: %d\n", devProp.multiProcessorCount);
		printf("  - Clock: %d MHz\n", devProp.clockRate/1000);
		printf("  - Max thread dim: dim3(%d, %d, %d)\n", devProp.maxThreadsDim[0], devProp.maxThreadsDim[1], devProp.maxThreadsDim[2]);
		printf("  - Warp size: %d\n", devProp.warpSize);
		printf("  - Regs/block: %d\n", devProp.regsPerBlock);
		printf("  - Regs/SM: %d\n", devProp.regsPerMultiprocessor);
		printf("  - Shared mem/Block: %d KB\n", devProp.sharedMemPerBlock/1024);
		printf("  - Shared mem/SM: %d KB\n", devProp.sharedMemPerMultiprocessor/1024);
		printf("  - Constant mem: %d KB\n", devProp.totalConstMem/1024);
		printf("  - Cache L2: %d MB\n", devProp.l2CacheSize/(1024*1024));
	}

	return 0;
}
