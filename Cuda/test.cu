#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define SIZE 1024

__global__ void vectorAdd(int *a, int *b, int *c, int n)
{
	int i = threadIdx.x;

	if(i<n)
		c[i]=a[i]+b[i];
}

int main()
{
	int *a, *b, *c;

	int id = hipGetDevice(&id);
	printf("GPU id: %d\n",id);

	hipMallocManaged(&a, SIZE*sizeof(int));
	hipMallocManaged(&b, SIZE*sizeof(int));
	hipMallocManaged(&c, SIZE*sizeof(int));

	for(int i=0;i<SIZE;i++)
	{
		a[i]=i;
		b[i]=i;
		c[i]=0;
	}

	// Start sending data to gpu
	hipMemPrefetchAsync(a, SIZE, id);
	hipMemPrefetchAsync(b, SIZE, id);
	vectorAdd<<<1, SIZE>>>(a,b,c, SIZE);
	hipDeviceSynchronize();
	
	// Send data back to cpu
	hipMemPrefetchAsync(c, SIZE, hipCpuDeviceId);

	for(int i=0;i<10;i++)
		printf("c[%d]=%d\n",i,c[i]);

	hipFree(a);
	hipFree(b);
	hipFree(c);

	return 0;
}
