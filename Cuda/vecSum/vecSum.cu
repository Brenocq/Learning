// nvcc vecSum.cu -o a && ./a

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 50

__global__ void vectorAddKernel(float* a, float* b, float* c, int n)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;

	if(i<n) c[i]=a[i]+b[i];
}

int main()
{
	float *d_a, *d_b, *d_c;
	float *h_a, *h_b, *h_c;

	int id = hipGetDevice(&id);
	printf("GPU id: %d\n",id);

	// Alloc memory CPU
	h_a = (float*)malloc(SIZE*sizeof(float));
	h_b = (float*)malloc(SIZE*sizeof(float));
	h_c = (float*)malloc(SIZE*sizeof(float));

	// Alloc memory GPU
	hipMalloc((void**)&d_a, SIZE*sizeof(float));
	hipMalloc((void**)&d_b, SIZE*sizeof(float));
	hipMalloc((void**)&d_c, SIZE*sizeof(float));

	// Initialize vectors
	for(int i=0;i<SIZE;i++)
	{
		h_a[i]=i;
		h_b[i]=i;
		h_c[i]=0;
	}

	// Copy from CPU to GPU
	hipMemcpy(d_a, h_a, SIZE*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, SIZE*sizeof(float), hipMemcpyHostToDevice);

	vectorAddKernel<<<ceil(SIZE/256.0), 256>>>(d_a,d_b,d_c, SIZE);
	
	// Copy from GPU to CPU
	hipMemcpy(h_c, d_c, SIZE*sizeof(float), hipMemcpyDeviceToHost);

	// Print
	for(int i=0;i<SIZE;i++)
		printf("c[%d]=%f\n",i,h_c[i]);

	// Free
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	free(h_a);
	free(h_b);
	free(h_c);

	return 0;
}
